
#include <hip/hip_runtime.h>
// #include "waveKernels.cuh"

// // complex math functions used in the kernels (GPU)
// ////////////////////////////////////////////////////////////////////////////////////////
// __device__ float2 conjugate(float2 arg) { return make_float2(arg.x, -arg.y);
// }

// __device__ float2 complex_exp(float arg) {
//   return make_float2(cosf(arg), sinf(arg));
// }

// __device__ float2 complex_add(float2 a, float2 b) {
//   return make_float2(a.x + b.x, a.y + b.y);
// }

// __device__ float2 complex_mult(float2 ab, float2 cd) {
//   return make_float2(ab.x * cd.x - ab.y * cd.y, ab.x * cd.y + ab.y * cd.x);
// }
// ////////////////////////////////////////////////////////////////////////////////////////

// // Kernels
// ////////////////////////////////////////////////////////////////////////////////////////

// // generate wave heightfield at time t based on initial heightfield and
// // dispersion relationship
// __global__ void generateTimeSpectrumKernel(float2 *h0, float2 *ht,
//                                            unsigned int in_width,
//                                            unsigned int out_width,
//                                            unsigned int out_height, float t,
//                                            float L) {
//   // global x,y based on cuda structure hierarchy
//   unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;  // go to x'th
//   column unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;  // go to
//   y'th row unsigned int in_index = y * in_width + x;  // go to y'th row, then
//   x'th column unsigned int in_mindex =
//       (out_height - y) * in_width + (out_width - x);  // mirrored
//   unsigned int out_index = y * out_width + x;

//   if ((x < out_width) && (y < out_height)) {
//     /*
//                     calculate wave vector k: -N/2 <= n,m < N/2,
//                     kx = 2*pi*n/L, ky = 2*pi*m/L
//     */
//     float2 k;
//     k.x = (-(int)out_width / 2.0f + x) * (2.0f * CUDART_PI_F / L);
//     k.y = (-(int)out_width / 2.0f + y) * (2.0f * CUDART_PI_F / L);

//     // dispersion relationship: w(k) = sqrt(g*|k|)
//     float k_len = sqrt(k.x * k.x + k.y * k.y);
//     float w = sqrtf(CUDART_GRAVITY_F * k_len);

//     float2 h0_k = h0[in_index];
//     float2 h0_mk = h0[in_mindex];

//     // output frequency-space complex values
//     ht[out_index] =
//         complex_add(complex_mult(h0_k, complex_exp(w * t)),
//                     complex_mult(conjugate(h0_mk), complex_exp(-w * t)));
//   }
// }

// // update height map values based on output of FFT
// __global__ void updateHeightmapKernel(float *heightMap, float2 *ht,
//                                       unsigned int width) {
//   unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
//   unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
//   unsigned int i = y * width + x;

//   // sign correction for FFT where the sign is flipped by checkerboard
//   pattern float sign_correction = ((x + y) & 0x01) ? -1.0f : 1.0f;

//   heightMap[i] = ht[i].x * sign_correction;
// }

// // generate slope by partial differences in spatial domain
// __global__ void calculateSlopeKernel(float *h, float2 *slopeOut,
//                                      unsigned int width, unsigned int height)
//                                      {
//   unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
//   unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
//   unsigned int i = y * width + x;

//   float2 slope = make_float2(0.0f, 0.0f);

//   if ((x > 0) && (y > 0) && (x < width - 1) && (y < height - 1)) {
//     slope.x = h[i + 1] - h[i - 1];
//     slope.y = h[i + width] - h[i - width];
//   }

//   slopeOut[i] = slope;
// }

// // Round a / b to nearest higher integer value
// int cuda_iDivUp(int a, int b) { return (a + (b - 1)) / b; }

// // wrapper functions
// extern "C" void cudaGenerateTimeSpectrumKernel(float2 *d_h0, float2 *d_ht,
//                                                unsigned int in_width,
//                                                unsigned int out_width,
//                                                unsigned int out_height,
//                                                float TIME, float patchSize) {
//   dim3 block(8, 8, 1);
//   dim3 grid(cuda_iDivUp(out_width, block.x), cuda_iDivUp(out_height,
//   block.y),
//             1);
//   generateTimeSpectrumKernel<<<grid, block>>>(d_h0, d_ht, in_width,
//   out_width,
//                                               out_height, TIME, patchSize);
// }

// extern "C" void cudaUpdateHeightmapKernel(float *d_heightMap, float2 *d_ht,
//                                           unsigned int width,
//                                           unsigned int height) {
//   dim3 block(8, 8, 1);
//   dim3 grid(cuda_iDivUp(width, block.x), cuda_iDivUp(height, block.y), 1);
//   updateHeightmapKernel<<<grid, block>>>(d_heightMap, d_ht, width);
// }

// extern "C" void cudaCalculateSlopeKernel(float *hptr, float2 *slopeOut,
//                                          unsigned int width,
//                                          unsigned int height) {
//   dim3 block(8, 8, 1);
//   dim3 grid(cuda_iDivUp(width, block.x), cuda_iDivUp(height, block.y), 1);
//   calculateSlopeKernel<<<grid, block>>>(hptr, slopeOut, width, height);
// }
